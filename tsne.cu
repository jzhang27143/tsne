#include <hip/hip_runtime.h>
#include <getopt.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>

#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>

#include "kernels/gradients.h"
#include "kernels/perplexity_search.h"
#include "kernels/utils.h"

void usage(const char *progname) {
    printf("Program Options:\n");
    printf("  -D  --dataset     <FILENAME>   Input file containing the raw feature data\n");
    printf("  -d  --nn_dists    <FILENAME>   Input file with L2 distances between nearest neighbors\n");
    printf("  -i  --nn_index    <FILENAME>   Input file identifying indices of nearest neighbors\n");
    printf("  -k  --k           <INT>        Number of nearest neighbors per point\n");
    printf("  -p  --perplexity  <FLOAT>      Perplexity target for variance initialization\n");
    printf("  -e  --epsilon     <FLOAT>      Convergence threshold for perplexity search\n");
    printf("  -T  --num_iters   <INT>        Number of gradient descent iterations\n");
    printf("  -h  --help                     This message\n");
}

template <class T>
thrust::device_vector<T> load_data(std::string filename, T (*convert)(const std::string&)) {
    // May need device_malloc for MNIST
    thrust::device_vector<T> vec;
    std::ifstream in_file(filename.c_str());
    if (in_file.is_open()) {
        std::string line, token;
        while (std::getline(in_file, line)) {
            std::stringstream ss(line);
            while (std::getline(ss, token, ',')) {
                vec.push_back(convert(token));
            }
        }
        in_file.close();
    }
    else {
        printf("Could not open file %s\n", filename.c_str());
        exit(1);
    }
    return vec;
}

int stoi(const std::string& s) {
    return std::stoi(s);
}

float stof(const std::string& s) {
    return std::stof(s);
}

void dump_final_embeds(std::string dataset_fname, int num_points,
                       thrust::host_vector<float> &embed_x,
                       thrust::host_vector<float> &embed_y) {
    
    std::string input_fname(dataset_fname);

    input_fname = input_fname.substr(5, input_fname.length() - 5 - 4);
    std::string output_fname = "./output_" + input_fname + ".txt";
    std::cout << output_fname << "\n";
    FILE *output = fopen(const_cast<char*>(output_fname.c_str()), "w");
    if (!output) {
        std::cout << "Unable to create file\n";
    }
    
    for (int i = 0; i < num_points; i++) {
        fprintf(output, "%.6f %.6f\n", embed_x[i], embed_y[i]);
    }

    fclose(output);
}

int main(int argc, char **argv) {

    // Parse commandline options
    int opt;
    static struct option long_options[] = {
        {"dataset", 1, 0, 'D'},
        {"nn_dists", 1, 0, 'd'},
        {"nn_index", 1, 0, 'i'},
        {"k", 1, 0, 'k'},
        {"perplexity", 1, 0, 'p'},
        {"theta", 1, 0, 't'},
        {"epsilon", 1, 0, 'e'},
        {"num_iters", 1, 0, 'T'},
        {"help", 0, 0, 'h'},
        {0, 0, 0, 0}
    };

    int k;
    int num_iters = 1000;
    float perplexity_target = 30.f;
    float epsilon = 1e-4;
    float theta = 0.5;
    std::string dataset_fname;
    std::string dists_fname;
    std::string index_fname;

    while ((opt = getopt_long(argc, argv, "D:d:i:k:p:t:e:T:h", long_options, NULL)) != EOF) {
        switch (opt) {
        case 'D':
            dataset_fname = optarg;
            break;
        case 'd':
            dists_fname = optarg;
            break;
        case 'i':
            index_fname = optarg;
            break;
        case 'k':
            k = atoi(optarg);
            break;
        case 'p':
            perplexity_target = atof(optarg);
            break;
        case 't':
            theta = atof(optarg);
            break;
        case 'e':
            epsilon = atof(optarg);
            break;
        case 'T':
            num_iters = atoi(optarg);
            break;
        case 'h':
        default:
            usage(argv[0]);
            return 1;
        }
    }

    thrust::device_vector<int> nn_index = load_data(index_fname, &stoi);
    thrust::device_vector<float> nn_dists = load_data(dists_fname, &stof);
    thrust::device_vector<float> dataset = load_data(dataset_fname, &stof);
    int num_points = nn_index.size() / k;

    thrust::device_vector<float> pij(num_points * k);
    thrust::device_vector<float> pij_sym(num_points * k);
    search_perplexity(pij, nn_dists, perplexity_target, epsilon, num_points, k);
    symmetrize_matrix(pij_sym, pij, nn_index, num_points, k);
    
    // Initialize 2D points
    thrust::device_vector<float> embed_x(num_points);
    thrust::device_vector<float> embed_y(num_points);
    initialize_points(embed_x, embed_y, num_points);

    // Initialize attractive and repulsive forces
    thrust::device_vector<float> grad_attract_x(num_points);
    thrust::device_vector<float> grad_attract_y(num_points);
    thrust::device_vector<float> grad_repulsive_x(num_points);
    thrust::device_vector<float> grad_repulsive_y(num_points);

    // Initialize parameters for t-SNE
    thrust::device_vector<float> old_forces_x(num_points, 0.f);
    thrust::device_vector<float> old_forces_y(num_points, 0.f);
    thrust::device_vector<float> gains_x(num_points, 1.f);
    thrust::device_vector<float> gains_y(num_points, 1.f);
    
    float learning_rate = 200.f;
    float momentum = 0.5f;
    float exaggeration = 12.f;
    
    for (int t = 0; t < num_iters; t++) {

        std::cout << "Iteration: " << t << std::endl;
        compute_attractive_forces(pij_sym, embed_x, embed_y, nn_index,
                                  grad_attract_x, grad_attract_y, num_points, k);
        compute_repulsive_forces(embed_x, embed_y, grad_repulsive_x, grad_repulsive_y,
                                 num_points, theta);
        if (t > 250) {
            momentum = 0.8f;
            exaggeration = 1.f;
        }
        apply_forces(embed_x, embed_y, gains_x, gains_y, old_forces_x, old_forces_y,
                     grad_attract_x, grad_attract_y,
                     grad_repulsive_x, grad_repulsive_y,
                     learning_rate, momentum, exaggeration, num_points);
    }
    /*
    thrust::host_vector<float> host_x(9);
    thrust::host_vector<float> host_y(9);
    host_x[0] = 0.f;
    host_x[1] = 0.f;
    host_x[2] = 0.f;
    host_x[3] = 1.f;
    host_x[4] = 1.f;
    host_x[5] = 1.f;
    host_x[6] = 2.f;
    host_x[7] = 2.f;
    host_x[8] = 2.f;

    host_y[0] = 0.f;
    host_y[1] = 1.f;
    host_y[2] = 2.f;
    host_y[3] = 0.f;
    host_y[4] = 1.f;
    host_y[5] = 2.f;
    host_y[6] = 0.f;
    host_y[7] = 1.f;
    host_y[8] = 2.f;
 

    thrust::device_vector<float> device_x = host_x;
    thrust::device_vector<float> device_y = host_y;   
    thrust::device_vector<float> grad_repulsive_x(9, 0.f);
    thrust::device_vector<float> grad_repulsive_y(9, 0.f);  
    
    for (int i = 0; i < 9; i++) {
        std::cout << grad_repulsive_x[i] << " " << grad_repulsive_y[i] << std::endl;
    }  

    compute_repulsive_forces(device_x, device_y, grad_repulsive_x, grad_repulsive_y, 9, theta);
    thrust::host_vector<float> host_repulsive_x(9);
    thrust::host_vector<float> host_repulsive_y(9);

    thrust::copy(grad_repulsive_x.begin(), grad_repulsive_x.end(), host_repulsive_x.begin());
    thrust::copy(grad_repulsive_y.begin(), grad_repulsive_y.end(), host_repulsive_y.begin());

    for (int i = 0; i < 9; i++) {
        std::cout << host_repulsive_x[i] << " " << host_repulsive_y[i] << std::endl;
    }
    */
    thrust::host_vector<float> host_embed_x(num_points);
    thrust::host_vector<float> host_embed_y(num_points);
    thrust::copy(embed_x.begin(), embed_x.end(), host_embed_x.begin());
    thrust::copy(embed_y.begin(), embed_y.end(), host_embed_y.begin());

    dump_final_embeds(dataset_fname, num_points, host_embed_x, host_embed_y);         
    return 0;
}
