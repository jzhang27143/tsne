#include "hip/hip_runtime.h"
#include "utils.h"
#include "cfloat"
#include "math.h"
#include "random"
#include "stdio.h"
#include "thrust/device_vector.h"
#include <vector>

__global__ void kernel_symmetrize_matrix(volatile float *__restrict__ pij_sym,
                                         const float *__restrict__ pij_unsym,
                                         const int *__restrict__ nn_indices, 
                                         int num_points, int k) {
    register int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index >= k * num_points) {
        return;
    }

    int i = index / k;
    int j = nn_indices[index];

    pij_sym[index] = pij_unsym[index];
    for (int idx = j*k; idx < j*k + k; idx++) {
        if (nn_indices[idx] == i) {
            pij_sym[index] += pij_unsym[idx];
        }
    }
    
    pij_sym[index] /= (2 * num_points);
}

void symmetrize_matrix(thrust::device_vector<float> &pij_unsym,
                       thrust::device_vector<float> &pij_sym,
                       thrust::device_vector<int> &nn_indices,
                       int num_points, int k) {
    const int BLOCKSIZE = 1024;
    const int NBLOCKS = (num_points * k + BLOCKSIZE - 1) / BLOCKSIZE;

    kernel_symmetrize_matrix<<<NBLOCKS, BLOCKSIZE>>>(
        thrust::raw_pointer_cast(pij_unsym.data()), 
        thrust::raw_pointer_cast(pij_sym.data()),
        thrust::raw_pointer_cast(nn_indices.data()),
        num_points, k
    ); 
}

void initialize_points(thrust::device_vector<float2> &ys, int num_points) {
    thrust::host_vector<float2> host_ys(num_points);

    std::default_random_engine generator(15618);
    std::normal_distribution<float> norm_dist(0.0, 0.0001);
    for (int i = 0; i < host_ys.size(); i++) {
        host_ys[i].x = norm_dist(generator);
        host_ys[i].y = norm_dist(generator);
    }

    thrust::copy(host_ys.begin(), host_ys.end(), ys.begin());   

}
