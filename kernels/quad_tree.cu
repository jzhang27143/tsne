#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "quad_tree.h"
#include "exclusiveScan.cu_inl"

__device__ float2 find_box_center(QuadTreeNode_t *n) {
    float center_x = 0.5f * (n->top_left.x + n->bottom_right.x);
    float center_y = 0.5f * (n->top_left.y + n->bottom_right.y);
    return make_float2(center_x, center_y);
}

__device__ float find_box_width(QuadTreeNode_t *n) {
    return n->bottom_right.x - n->top_left.x; 
}

__device__ float find_box_height(QuadTreeNode_t *n) {
    return n->bottom_right.y - n->top_left.y;
}

__global__ void kernel_center_of_mass(QuadTreeNode_t *nodes, const int max_depth) {
    register int tid = blockIdx.x * blockDim.x + threadIdx.x;
    register int num_threads = blockDim.x * gridDim.x;

    for (int d = max_depth - 1; d >= 0; d--) {
        register int start = ((1 << (2 * d)) - 1) / 3;
        register int end = ((1 << (2 * d + 2)) - 1) / 3;

        for (int i = start + tid; i < end; i += num_threads) {
            QuadTreeNode_t *node = &nodes[i];
            if (node->is_node && !node->is_leaf) {
                float x = 0.f, y = 0.f;
                int N = node->num_points;
                QuadTreeNode_t *top_left = &nodes[4 * i + 1];
                QuadTreeNode_t *top_right = &nodes[4 * i + 2];
                QuadTreeNode_t *bottom_left = &nodes[4 * i + 3];
                QuadTreeNode_t *bottom_right = &nodes[4 * i + 4];
                if (top_left->is_node) {
                    x += (&nodes[4 * i + 1])->center_of_mass.x * (&nodes[4 * i + 1])->num_points;
                    y += (&nodes[4 * i + 1])->center_of_mass.y * (&nodes[4 * i + 1])->num_points;
                }               
                
                if (top_right->is_node) {
                    x += (&nodes[4 * i + 2])->center_of_mass.x * (&nodes[4 * i + 2])->num_points;
                    y += (&nodes[4 * i + 2])->center_of_mass.y * (&nodes[4 * i + 2])->num_points;
                }
 
                if (bottom_left->is_node) {
                    x += (&nodes[4 * i + 3])->center_of_mass.x * (&nodes[4 * i + 3])->num_points;
                    y += (&nodes[4 * i + 3])->center_of_mass.y * (&nodes[4 * i + 3])->num_points;
                }

                if (bottom_right->is_node) {
                    x += (&nodes[4 * i + 4])->center_of_mass.x * (&nodes[4 * i + 4])->num_points;
                    y += (&nodes[4 * i + 4])->center_of_mass.y * (&nodes[4 * i + 4])->num_points;
                }
                node->center_of_mass = make_float2(x / N, y / N);
            }
        }
        __syncthreads();
    }
}

__global__ void kernel_build_quadtree(float *__restrict__ embed_x_in,
                                      float *__restrict__ embed_y_in,
                                      float *__restrict__ embed_x_out,
                                      float *__restrict__ embed_y_out,
                                      QuadTreeNode_t *nodes,
                                      int remaining_depth,
                                      int node_idx) {

    extern __shared__ uint smem[];
    int root_node_idx = node_idx + blockIdx.x;    

    int start = nodes[root_node_idx].start;
    int end = nodes[root_node_idx].end;

    register int num_threads = blockDim.x;
    uint *bucket_counts_tl = smem;
    uint *bucket_counts_tr = &smem[num_threads];
    uint *bucket_counts_bl = &smem[2 * num_threads];
    uint *bucket_counts_br = &smem[3 * num_threads];
    uint *ex_scan_tl = &smem[4 * num_threads];
    uint *ex_scan_tr = &smem[5 * num_threads];
    uint *ex_scan_bl = &smem[6 * num_threads];
    uint *ex_scan_br = &smem[7 * num_threads];
    uint *ex_scan_scratch = &smem[8 * num_threads];

    // register int tid = blockIdx.x * blockDim.x + threadIdx.x;
    register int tid = threadIdx.x;
    register int num_points = end - start;
    if (num_points == 0) {
        return;
    }

    QuadTreeNode_t *node = &nodes[root_node_idx];
    if (tid == 0) {
        node->num_points = num_points;
        node->is_node = true;
    }

    if (num_points == 1 || remaining_depth == 0) {
        if (tid == 0) {
            float x_center_of_mass = 0.f;
            float y_center_of_mass = 0.f;
            for (int i = start; i < end; i++) {
                x_center_of_mass += embed_x_in[i];
                y_center_of_mass += embed_y_in[i];
            }

            x_center_of_mass /= num_points;
            y_center_of_mass /= num_points;
            node->center_of_mass = make_float2(x_center_of_mass, y_center_of_mass);
            node->is_leaf = true;
        }
        return;
    }

    float2 box_center = find_box_center(node);
    int num_top_left = 0;
    int num_top_right = 0;
    int num_bottom_left = 0;
    int num_bottom_right = 0;

    // Step 1: Each thread counts the number of points per bucket
    for (int i = start + tid; i < end; i += num_threads) {
        float x = embed_x_in[i];
        float y = embed_y_in[i];
        if (x <= box_center.x && y <= box_center.y) {
            num_top_left++;
        }
        else if (x > box_center.x && y <= box_center.y) {
            num_top_right++;
        }
        else if (x <= box_center.x && y > box_center.y) {
            num_bottom_left++;
        }
        else {
            num_bottom_right++;
        }
    }

    bucket_counts_tl[tid] = num_top_left;
    bucket_counts_tr[tid] = num_top_right;
    bucket_counts_bl[tid] = num_bottom_left;
    bucket_counts_br[tid] = num_bottom_right;

    // Step 2: Perform Exclusive Scan for each bucket
    sharedMemExclusiveScan(tid, bucket_counts_tl, ex_scan_tl, ex_scan_scratch, num_threads);
    sharedMemExclusiveScan(tid, bucket_counts_tr, ex_scan_tr, ex_scan_scratch, num_threads);
    sharedMemExclusiveScan(tid, bucket_counts_bl, ex_scan_bl, ex_scan_scratch, num_threads);
    sharedMemExclusiveScan(tid, bucket_counts_br, ex_scan_br, ex_scan_scratch, num_threads);
    __syncthreads();

    // Step 3: Compute offsets for reordered indices
    int offset_tr = start + ex_scan_tl[num_threads - 1] + bucket_counts_tl[num_threads - 1];
    int offset_bl = offset_tr + ex_scan_tr[num_threads - 1] + bucket_counts_tr[num_threads - 1];
    int offset_br = offset_bl + ex_scan_bl[num_threads - 1] + bucket_counts_bl[num_threads - 1];
    __syncthreads();

    ex_scan_tl[tid] += start;
    ex_scan_tr[tid] += offset_tr;
    ex_scan_bl[tid] += offset_bl;
    ex_scan_br[tid] += offset_br;
    __syncthreads();

    // Step 4: Fill in points in bucket sort order
    int fill_idx_tl = ex_scan_tl[tid];
    int fill_idx_tr = ex_scan_tr[tid];
    int fill_idx_bl = ex_scan_bl[tid];
    int fill_idx_br = ex_scan_br[tid];

    for (int i = start + tid; i < end; i += num_threads) {
        float x = embed_x_in[i];
        float y = embed_y_in[i];
        if (x <= box_center.x && y <= box_center.y) {
            embed_x_out[fill_idx_tl] = x;
            embed_y_out[fill_idx_tl++] = y;
        }
        else if (x > box_center.x && y <= box_center.y) {
            embed_x_out[fill_idx_tr] = x;
            embed_y_out[fill_idx_tr++] = y;
        }
        else if (x <= box_center.x && y > box_center.y) {
            embed_x_out[fill_idx_bl] = x;
            embed_y_out[fill_idx_bl++] = y;
        }
        else {
            embed_x_out[fill_idx_br] = x;
            embed_y_out[fill_idx_br++] = y;
        }
    }
    __syncthreads();

    if (tid == 0) {
        int smem_size = 10 * num_threads * sizeof(int);
        float2 top_left = nodes[root_node_idx].top_left;
        float2 bottom_right = nodes[root_node_idx].bottom_right;        

        nodes[4 * root_node_idx + 1].top_left = top_left;
        nodes[4 * root_node_idx + 1].bottom_right = box_center;
        nodes[4 * root_node_idx + 2].top_left = make_float2(box_center.x, top_left.y);
        nodes[4 * root_node_idx + 2].bottom_right = make_float2(bottom_right.x, box_center.y);
        nodes[4 * root_node_idx + 3].top_left = make_float2(top_left.x, box_center.y);
        nodes[4 * root_node_idx + 3].bottom_right = make_float2(box_center.x, bottom_right.y);
        nodes[4 * root_node_idx + 4].top_left = box_center;
        nodes[4 * root_node_idx + 4].bottom_right = bottom_right;

        nodes[4 * root_node_idx + 1].start = start;
        nodes[4 * root_node_idx + 1].end = ex_scan_tr[0];
        nodes[4 * root_node_idx + 2].start = ex_scan_tr[0];
        nodes[4 * root_node_idx + 2].end = ex_scan_bl[0];
        nodes[4 * root_node_idx + 3].start = ex_scan_bl[0];
        nodes[4 * root_node_idx + 3].end = ex_scan_br[0];
        nodes[4 * root_node_idx + 4].start = ex_scan_br[0];
        nodes[4 * root_node_idx + 4].end = end;

        // Launching kernel for all 4 quadrants
        kernel_build_quadtree<<<4, num_threads, smem_size>>>(
            embed_x_out, embed_y_out, embed_x_in, embed_y_in,
            nodes, remaining_depth - 1, 4 * root_node_idx + 1
        );
    }
}
